﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "./components/headers/Ray.h"

using namespace components;

int main()
{
    Vector3D v1 = Vector3D(0, 0, 0);
    Vector3D v2 = Vector3D(1, 2, 3);
    Ray ray = Ray(v1, v2);

    Vector3D resultant = ray.evaluate(5);

    printf("Resultant:\n\tx: %lf\n\ty: %lf\n\tz: %lf", resultant.x, resultant.y, resultant.z);

    return 0;
}