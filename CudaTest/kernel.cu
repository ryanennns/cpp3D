﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "./Vector3D.cpp"

int main()
{
    Vector3D vector1(5.0, 3.0, 10.0);
    Vector3D vector2(2.0, 7.0, 1.0);
    Vector3D::debug(vector1, vector2);

    return 0;
}