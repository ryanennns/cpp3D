﻿//#include "hip/hip_runtime.h"
//#include ""

#include "./src/include/Object.h"
#include "./src/include/Ray.h"
#include "./src/include/Rgb.h"
#include "./src/include/Scene.h"
#include "./src/include/Sphere.h"
#include "./src/include/Surface.h"
#include "./src/include/Triangle.h"
#include "./src/include/Vector3D.h"
#include "./src/include/ViewDriver.h"
#include "./src/include/ViewPort.h"

int main()
{
    return 0;
}