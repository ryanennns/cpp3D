﻿//#include "hip/hip_runtime.h"
//#include ""

#include <stdio.h>
#include "./src/components/headers/Ray.h"
#include "./src/environment/resources/shapes/sphere.h"

using namespace components;
using namespace environment;

int main()
{
    Sphere sphere = Sphere(Vector3D(5,5,0), 1);
    Ray ray = Ray(Vector3D(0,0,0), Vector3D(0,0,1));

    printf("Intersects: %d", sphere.rayIntersect(ray, 0));

    return 0;
}