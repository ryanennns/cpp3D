﻿//#include "hip/hip_runtime.h"
//#include ""

#include "./src/include/Object.h"
#include "./src/include/Ray.h"
#include "./src/include/Rgb.h"
#include "./src/include/Scene.h"
#include "./src/include/Sphere.h"
#include "./src/include/Surface.h"
#include "./src/include/Triangle.h"
#include "./src/include/Vector3D.h"
#include "./src/include/ViewDriver.h"
#include "./src/include/ViewPort.h"
#include "./src/include/Light.h"
#include "./src/include/HitDetection.h"
#include "./src/include/RaySurfaceIntersection.h"

int main()
{
    return 0;
}